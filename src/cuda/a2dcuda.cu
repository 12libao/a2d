/**
 * CUDA functionalities: unified memory allocation, kernel wrapper, etc.
 */

#include "a2dcuda.h"

int cuda_malloc(void **ptr_addr, std::size_t size) {
  hipError_t ret = hipMallocManaged(ptr_addr, size);
  return static_cast<int>(ret);
}

int cuda_free(void *ptr) {
  hipError_t ret = hipFree(ptr);
  return static_cast<int>(ret);
}
